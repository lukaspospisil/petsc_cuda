#include "hip/hip_runtime.h"
/* include petsc */
#include "petsc.h"
#include "mpi.h"

/* for measuring computation time */
#include "include/timer.h"
#ifdef USE_CUDA
	/* some cuda helpers */
	#include "include/cuda_stuff.h"
#endif

#define X_SIZE 1e6
#define N_TRIALS 1000
#define PRINT_VECTOR_CONTENT 0

int main( int argc, char *argv[] )
{
	/* error handling */
	PetscErrorCode ierr; 
	
	/* initialize Petsc */
	PetscInitialize(&argc,&argv,PETSC_NULL,PETSC_NULL);

	/* warm up GPU - call empty kernel (see include/cuda_stuff.h) */
	#ifdef USE_CUDA
		warm_up_cuda();
	#endif

	/* problem dimensions */
	int n = X_SIZE; /* length of vectors */
	int ntrials = N_TRIALS; /* number of trials (to provide average time) */
	
	/* print info about benchmark */
	ierr = PetscPrintf(PETSC_COMM_WORLD,"This is LOAD test.\n"); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD," - n          : %d\t\t(length of vectors)\n",n); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD," - ntrials    : %d\t\t(number of trials)\n",ntrials); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"-------------------------------\n"); CHKERRQ(ierr);
	
	Timer mytimer;
	mytimer.restart();
	mytimer.start();

	/* create vector x */
	Vec x;
	ierr = VecCreate(PETSC_COMM_WORLD,&x); CHKERRQ(ierr);
	ierr = VecSetSizes(x,PETSC_DECIDE,n); CHKERRQ(ierr);
	#ifdef USE_CUDA
		/* if we are using CUDA, it is a good idea to compute on GPU */
		ierr = VecSetType(x, VECMPICUDA); CHKERRQ(ierr);
	#else
		ierr = VecSetType(x, VECMPI); CHKERRQ(ierr);
	#endif
	ierr = VecSetFromOptions(x); CHKERRQ(ierr);

	/* some values */
	ierr = VecSet(x,1.0); CHKERRQ(ierr);

	mytimer.stop();
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- vector prepared in: %f s\n",mytimer.get_value_last()); CHKERRQ(ierr);

	/* maybe print the content of the vector ? */
	if(PRINT_VECTOR_CONTENT){
		ierr = PetscPrintf(PETSC_COMM_WORLD,"\n- Vector content: -------------\n"); CHKERRQ(ierr);
		ierr = VecView(x, PETSC_VIEWER_STDOUT_WORLD); CHKERRQ(ierr);
		ierr = PetscPrintf(PETSC_COMM_WORLD,"-------------------------------\n"); CHKERRQ(ierr);
	}

	/* save vector */
	mytimer.start();
	
	/* prepare viewer to save to file, save the vector and destroy viewer */
	PetscViewer mviewer;
	ierr = PetscViewerCreate(PETSC_COMM_WORLD, &mviewer); CHKERRQ(ierr);
	ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD, "my_vector.bin", FILE_MODE_WRITE, &mviewer); CHKERRQ(ierr);
	ierr = VecView(x, mviewer); CHKERRQ(ierr);
	ierr = PetscViewerDestroy(&mviewer); CHKERRQ(ierr);
	mytimer.stop();
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- vector saved in: %f s\n",mytimer.get_value_last()); CHKERRQ(ierr);

	/* destroy saved vector */
	ierr = VecDestroy(&x); CHKERRQ(ierr);

	/* prepare new vector */
	Vec y;
	ierr = VecCreate(PETSC_COMM_WORLD,&y); CHKERRQ(ierr);
	#ifdef USE_CUDA
		/* if we are using CUDA, it is a good idea to compute on GPU */
		ierr = VecSetType(y, VECMPICUDA); CHKERRQ(ierr);
	#else
		ierr = VecSetType(y, VECMPI); CHKERRQ(ierr);
	#endif
	ierr = VecSetFromOptions(y); CHKERRQ(ierr);
	
	/* load data from saved values */
	PetscViewer mviewer2;
	ierr = PetscViewerCreate(PETSC_COMM_WORLD, &mviewer2); CHKERRQ(ierr);
	ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD ,"my_vector.bin", FILE_MODE_READ, &mviewer2); CHKERRQ(ierr);
	ierr = VecLoad(y, mviewer2); CHKERRQ(ierr);
	ierr = PetscViewerDestroy(&mviewer2); CHKERRQ(ierr);
	
	/* HERE is the question - do I have to call this function to be sure that operations with this vector will be performed on GPU? */
	#ifdef USE_CUDA
		/* make sure that we are computing on GPU */
		ierr = VecCUDACopyToGPU(y); CHKERRQ(ierr);
	#endif	
	
	/* compute sum */
	double mysum = -1.0;
	
	mytimer.start();
	for(int i=0;i<N_TRIALS;i++){
		ierr = VecSum(y,&mysum); CHKERRQ(ierr);
	}
	mytimer.stop();

	double theory_sum = n;
	ierr = PetscPrintf(PETSC_COMM_WORLD,"\n- SUM info: ----------------\n"); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- theoretical sum       : %f\n",theory_sum); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- computed sum          : %f\n",mysum); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- difference            : %g\n",std::abs(mysum-theory_sum)); CHKERRQ(ierr);

	ierr = PetscPrintf(PETSC_COMM_WORLD,"- total time            : %g s\n", mytimer.get_value_last()); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"- avg.  time            : %g s\n", mytimer.get_value_last()/(double)N_TRIALS); CHKERRQ(ierr);
	ierr = PetscPrintf(PETSC_COMM_WORLD,"-------------------------------\n"); CHKERRQ(ierr);

	/* destroy vector */
	ierr = VecDestroy(&y); CHKERRQ(ierr);

	/* finalize Petsc */
	PetscFinalize();

	return 0;
}


